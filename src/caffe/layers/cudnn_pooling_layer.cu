
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_pooling_layer.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<__half>*>& bottom,
    const vector<Blob<__half>*>& top) {
  const __half* bottom_data = bottom[0]->gpu_data();
  __half* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<__half>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<__half>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const __half* top_diff = top[0]->gpu_diff();
  const __half* top_data = top[0]->gpu_data();
  const __half* bottom_data = bottom[0]->gpu_data();
  __half* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNPoolingLayer);

}  // namespace caffe
#endif
