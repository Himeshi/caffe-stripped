#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/split_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void SplitLayer<Dtype>::Forward_gpu(const vector<Blob<__half>*>& bottom,
      const vector<Blob<__half>*>& top) {
  for (int i = 0; i < top.size(); ++i) {
    top[i]->ShareData(*bottom[0]);
  }
}

template <typename Dtype>
void SplitLayer<Dtype>::Backward_gpu(const vector<Blob<__half>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<__half>*>& bottom) {
  if (!propagate_down[0]) { return; }
  if (top.size() == 1) {
    caffe_copy(count_, top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff());
    return;
  }
  caffe_gpu_add(count_, top[0]->gpu_diff(), top[1]->gpu_diff(),
                bottom[0]->mutable_gpu_diff());
  // Add remaining top blob diffs.
  for (int i = 2; i < top.size(); ++i) {
    const __half* top_diff = top[i]->gpu_diff();
    __half* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_axpy(count_, Dtype(1.), top_diff, bottom_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(SplitLayer);

}  // namespace caffe
