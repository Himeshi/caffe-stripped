
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layers/cudnn_softmax_layer.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<__half>*>& bottom,
    const vector<Blob<__half>*>& top) {
  const __half* bottom_data = bottom[0]->gpu_data();
  __half* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<__half>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<__half>*>& bottom) {
  if (propagate_down[0]) {
    const __half* top_data = top[0]->gpu_data();
    const __half* top_diff = top[0]->gpu_diff();
    const __half* bottom_data = bottom[0]->gpu_data();
    __half* bottom_diff = bottom[0]->mutable_gpu_diff();

    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
          CUDNN_SOFTMAX_MODE_CHANNEL,
          cudnn::dataType<Dtype>::one,
          top_desc_, top_data, top_desc_, top_diff,
          cudnn::dataType<Dtype>::zero,
          bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
