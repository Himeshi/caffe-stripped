#include "hip/hip_runtime.h"
#include "caffe/fp16.hpp"
#include "caffe/fp16.cuh"
namespace caffe {

void copy_posit_globals_to_gpu(int nbits, int esize, int useed, int useed_zeros, int posit_shift_amount,
  int maxrealexp, POSIT_TYPE maxrealp, POSIT_TYPE minrealp, POSIT_TYPE infp, float maxreal, float minreal) {
  hipMemcpyToSymbol(HIP_SYMBOL(_g_nbits_gpu), &nbits, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_esize_gpu), &esize, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_useed_gpu), &useed, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_useed_zeros_gpu), &useed_zeros, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_posit_shift_amount_gpu), &posit_shift_amount, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_maxrealexp_gpu), &maxrealexp, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_maxrealp_gpu), &maxrealp, sizeof(posit));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_minrealp_gpu), &minrealp, sizeof(posit));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_infp_gpu), &infp, sizeof(posit));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_maxreal_gpu), &maxreal, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_minreal_gpu), &minreal, sizeof(float));
}

__global__ void convert_to_fp16(const int n, float* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = fp32tofp16_gpu(in[index]);
  }
}

__global__ void convert_to_fp16(const int n, double* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp32tofp16_gpu(in[index]);
  }
}
__global__ void convert_to_float(const int n,  fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n,  fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
  
}

__global__ void outputweights(const int n, float* in) {
  CUDA_KERNEL_LOOP(index, n) {
printf("%d %f\n", index, in[index]);
  }
}

__global__ void outputweights(const int n, double* in) {
  CUDA_KERNEL_LOOP(index, n) {
printf("%d %f\n", index, in[index]);
  }
}

}
