#include "hip/hip_runtime.h"
#include "caffe/fp16.hpp"
#include "caffe/fp16.cuh"
namespace caffe {

__global__ void convert_to_fp16(const int n, float* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = fp32tofp16_gpu(in[index]);
  }
}

__global__ void convert_to_fp16(const int n, double* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp32tofp16_gpu(in[index]);
  }
}
__global__ void convert_to_float(const int n,  fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n,  fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
  
}

__global__ void outputweights(const int n, float* in) {
  CUDA_KERNEL_LOOP(index, n) {
printf("%d %f\n", index, in[index]);
  }
}

__global__ void outputweights(const int n, double* in) {
  CUDA_KERNEL_LOOP(index, n) {
printf("%d %f\n", index, in[index]);
  }
}

void print_gpu_float_array(const float* d_data, int size) {
	float *h_data;
	h_data = (float *) malloc(size * sizeof(float));
	hipMemcpy(h_data, d_data, size * sizeof(float), hipMemcpyDeviceToHost);
	printf("size = %d\n", size);
	int i;
	for (i = 0; i < size; i++) {
	    if(h_data[i] != 0.0)
		  printf("data[%d] = %f ", i, h_data[i]);
	}
	free(h_data);
}

void print_gpu_float_array(const double* d_data, int size) {
	double *h_data;
	h_data = (double *) malloc(size * sizeof(double));
	hipMemcpy(h_data, d_data, size * sizeof(double), hipMemcpyDeviceToHost);
	printf("size = %d\n", size);
	int i;
	for (i = 0; i < size; i++) {
	    if(h_data[i] != 0.0)
		  printf("data[%d] = %f ", i, h_data[i]);
	}
	free(h_data);
}

void print_gpu_fp16_array(const fp16* d_data, int size) {
	fp16 *h_data;
	h_data = (fp16 *) malloc(size * sizeof(fp16));
	hipMemcpy(h_data, d_data, size * sizeof(fp16), hipMemcpyDeviceToHost);
	printf("size = %d\n", size);
	int i;
	for (i = 0; i < size; i++) {
	    if(h_data[i] != 0)
		  printf("data[%d] = %f ", i, fp16tofp32(h_data[i]));
	}
	free(h_data);
}

}
