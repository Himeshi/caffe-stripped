#include "hip/hip_runtime.h"
#include "caffe/fp16.hpp"
#include "caffe/fp16.cuh"
namespace caffe {

void copy_posit_globals_to_gpu(int nbits, int esize, int useed, int useed_zeros, int posit_shift_amount,
  int maxrealexp, FP16_TYPE maxrealp, FP16_TYPE minrealp, FP16_TYPE infp, float maxreal, float minreal) {
  hipMemcpyToSymbol(HIP_SYMBOL(_g_nbits_gpu), &nbits, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_esize_gpu), &esize, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_useed_gpu), &useed, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_useed_zeros_gpu), &useed_zeros, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_posit_shift_amount_gpu), &posit_shift_amount, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_maxrealexp_gpu), &maxrealexp, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_maxrealp_gpu), &maxrealp, sizeof(fp16));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_minrealp_gpu), &minrealp, sizeof(fp16));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_infp_gpu), &infp, sizeof(fp16));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_maxreal_gpu), &maxreal, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(_g_minreal_gpu), &minreal, sizeof(float));
}

__global__ void convert_to_fp16(const int n, float* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = fp32tofp16_gpu(in[index]);
  }
}

__global__ void convert_to_fp16(const int n, double* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp32tofp16_gpu(in[index]);
  }
}
__global__ void convert_to_float(const int n,  fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n,  fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
  
}

__global__ void outputweights(const int n, float* in) {
  CUDA_KERNEL_LOOP(index, n) {
printf("%d %f\n", index, in[index]);
  }
}

__global__ void outputweights(const int n, double* in) {
  CUDA_KERNEL_LOOP(index, n) {
printf("%d %f\n", index, in[index]);
  }
}

}
