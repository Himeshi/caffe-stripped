#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"

#include "caffe/fp16.cuh"

namespace caffe {

template <typename Dtype>
__global__ void SGDUpdate(int N, FP16_TYPE* g, FP16_TYPE* h,
    Dtype momentum, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    g[i] = h[i] = fp32tofp16_gpu(momentum*fp16tofp32_gpu(h[i]) + local_rate*fp16tofp32_gpu(g[i]));
  }
}
template <typename Dtype>
void sgd_update_gpu(int N, FP16_TYPE* g, FP16_TYPE* h, Dtype momentum,
    Dtype local_rate) {
  SGDUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, momentum, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void sgd_update_gpu<float>(int, FP16_TYPE*, FP16_TYPE*, float, float);
template void sgd_update_gpu<double>(int, FP16_TYPE*, FP16_TYPE*, double, double);

}  // namespace caffe
